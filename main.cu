
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include <iomanip>

__global__ void reduce0(float* g_odata, float* g_idata, int n)
{
	extern __shared__ float sdata[];

	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

	sdata[tid] = (i<n)?g_idata[i]:0;
	__syncthreads();

	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{
		if (tid % (2 * s) == 0)
			sdata[tid] += sdata[tid + s];
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
	
}

__global__ void reduce1(float* g_odata, float* g_idata, int n)
{
	extern __shared__ float sdata[];

	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

	sdata[tid] = (i<n)?g_idata[i]:0;
	__syncthreads();

	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{
		unsigned int index = 2*s*tid;
		if (index < blockDim.x)
			sdata[tid] += sdata[tid + s];
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
	
}

float cpu_sum(float* h_in, float h_in_len)
{
	float total_sum = 0;
	for (int i = 0; i < h_in_len; ++i)
		total_sum = total_sum + h_in[i];

	return total_sum;
}

void gpu_sum(int whichKernel, int blocks, int threads, float* g_odata, float* g_idata, int n)
{
	int smem_size = (threads <= 32) ? 2*threads*sizeof(float) : threads * sizeof(float);
	switch (whichKernel)
	{
		case 0:
			reduce0<<<blocks, threads, smem_size>>>(g_odata, g_idata, n);
			break;
		case 1:
			reduce1<<<blocks, threads, smem_size>>>(g_odata, g_idata, n);
			break;
	}
	return;
}

int main()
{
	std::clock_t start;
	double duration;

	for (int len = 1<<10; len < 1<<30; len *=2)
	{
		std::cout << std::endl << len << std::endl;
		float* in;
		float* out;
		int blocksize = 1024;
	
		hipMallocManaged(&in, sizeof(float) * len);
		hipMallocManaged(&out, sizeof(float) * (len+blocksize-1)/blocksize);
	
		for (int i = 0; i < len; ++i)
			in[i] = i/100.0;
	
		std::cout.setf(std::ios::fixed,std::ios::floatfield);
		// Examine CPU time
		start = std::clock();
		// Call CPU sum here
		float cpu_out = cpu_sum(in, len);
		std::cout << cpu_out << std::endl;
		duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
		std::cout << "CPU time: " << duration << " s" << std::endl;
	
		//examine GPU time
		start = std::clock();
		// Call GPU sum here and sync
		gpu_sum(1, (len+blocksize-1)/blocksize, blocksize, out, in, len);
		hipDeviceSynchronize();
		float gpu_out = 0;
		for (int i = 0; i < (len+blocksize-1)/blocksize; ++i)
		{
			gpu_out += out[i];
		}
		std::cout << gpu_out << std::endl;
		duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
		std::cout << "GPU time: " << duration << " s" << std::endl;
	
		if (cpu_out-gpu_out < 1 && gpu_out-cpu_out < 1)
			std::cout << "wolf, tank, best match!" << std::endl;
	
		hipFree(in);
	}


	return 0;
}